#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "utils.cuh"

int main()
{
    int M, N, K;
    M = 4;
    N = 5;
    K = 6;

    srand(2019);
    // initialize host buffers
    helper::CBuffer<half> inputMatrix1, inputMatrix2;
    helper::CBuffer<float> outputMatrix;
    float alpha, beta;

    inputMatrix1.init(K * M, true);
    inputMatrix2.init(N * K, true);
    outputMatrix.init(N * M, true);

    bool tensor_core = false;

    // create cuda event handles
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // create cublas handle
    hipblasHandle_t cublas_handle;
    checkCudaErrors(
        hipblasCreate(&cublas_handle));

    int print_threshold = 12;
    if (M < print_threshold && N < print_threshold && K < print_threshold) {
        std::cout << "inputMatrix1:" << std::endl;
        helper::printMatrix(inputMatrix1.h_ptr_, K, M);
        std::cout << "inputMatrix2:" << std::endl;
        helper::printMatrix(inputMatrix2.h_ptr_, N, K);
        std::cout << "outputMatrix:" << std::endl;
        helper::printMatrix(outputMatrix.h_ptr_, N, M);
    }

    alpha = 1.f;
    beta = 0.f;

    // determin data type information for GemmEx()
    hipDataType TYPE_A, TYPE_B, TYPE_C;
    if (typeid(*inputMatrix1.h_ptr_) == typeid(float)) {
        TYPE_A = TYPE_B = HIP_R_32F;
    }
    else if (typeid(*inputMatrix1.h_ptr_) == typeid(half)) {
        TYPE_A = TYPE_B = HIP_R_16F;
    }
    else if (typeid(*inputMatrix1.h_ptr_) == typeid(int8_t)) {
        TYPE_A = TYPE_B = HIP_R_8I;
    }
    else {
        printf("Not supported precision\n");
        return -1;
    }

    if (typeid(*outputMatrix.h_ptr_) == typeid(float)) {
        TYPE_C = HIP_R_32F;
    }
    else if (typeid(*outputMatrix.h_ptr_) == typeid(int)) {
        TYPE_C = HIP_R_32I;
    }
    else {
        printf("Not supported precision\n");
        return -1;
    }

    // allocate GPU memory and copy the data
    inputMatrix1.cuda(true);
    inputMatrix2.cuda(true);
    outputMatrix.cuda(true);

    // enables tensorcore operation when it is possible
    // checkCudaErrors(
    //     hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH));

    hipEventRecord(start);
    checkCudaErrors(
        hipblasGemmEx(cublas_handle,
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        M, N, K,
                        &alpha,
                        inputMatrix1.d_ptr_, TYPE_A, M,
                        inputMatrix2.d_ptr_, TYPE_B, K,
                        &beta,
                        outputMatrix.d_ptr_, TYPE_C, M,
                        TYPE_C,
                        (tensor_core) ? CUBLAS_GEMM_DEFAULT_TENSOR_OP : HIPBLAS_GEMM_DEFAULT));
    hipEventRecord(stop);

    outputMatrix.copyToHost();

    if (M < print_threshold && N < print_threshold && K < print_threshold) {
        std::cout << "outputMatrix out:" << std::endl;
        helper::printMatrix(outputMatrix.h_ptr_, N, M);
    }

    // print out elapsed time
    float cudaElapsedTime;
    hipEventElapsedTime(&cudaElapsedTime, start, stop);
    std::cout << std::setw(4) << cudaElapsedTime << " ms" << std::endl;

    checkCudaErrors(
        hipblasDestroy(cublas_handle));

    return 0;
}
